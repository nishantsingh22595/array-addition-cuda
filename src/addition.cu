#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel to add two arrays
__global__ void addKernel(int* d_a, int* d_b, int* d_c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        d_c[index] = d_a[index] + d_b[index];
    }
}

void addArrays(int* h_a, int* h_b, int* h_c, int N) {
    int *d_a, *d_b, *d_c;

    size_t size = N * sizeof(int);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
